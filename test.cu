#include "hip/hip_runtime.h"
// test.cu
//
// demo of using thrust library to parse a large '|' delimited file.
//
// Build instructions at a Visual Studio 2010 Command Prompt: [CUDA 8.0 in the path here]
// nvcc -O3 -arch=sm_21 -lcuda test.cu -o test
// 
// Run with 'test.exe' <ENTER>

#include <iostream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <ctime>
#include "nvparse.h"

#ifdef _WIN64
#define atoll(S) _atoi64(S)
#include <windows.h>
#else
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#endif

int main()
{
    char * filename = "lineitem_small.tbl";

	std::clock_t start1 = std::clock();

    // get filesize
    FILE* f = fopen(filename, "r" );
    if (!f) {
        printf("Unable to open file %s \n",filename);
        exit(1);
    }
    fseek(f, 0, SEEK_END);
    long fileSize = ftell(f);
    fclose(f);

    // reserve filesize char vector on GPU
    thrust::device_vector<char> dev(fileSize);

#ifdef _WIN64
    // fast read of file using mapping
	HANDLE file = CreateFileA(filename, GENERIC_READ, FILE_SHARE_READ, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(file != INVALID_HANDLE_VALUE);

    HANDLE fileMapping = CreateFileMapping(file, NULL, PAGE_READONLY, 0, 0, NULL);
    assert(fileMapping != INVALID_HANDLE_VALUE);
 
    LPVOID fileMapView = MapViewOfFile(fileMapping, FILE_MAP_READ, 0, 0, 0);
    auto fileMapViewChar = (const char*)fileMapView;
    assert(fileMapView != NULL);

    thrust::copy(fileMapViewChar, fileMapViewChar+fileSize, dev.begin());
#else
    // non-Windows read of file via mapping
    struct stat sb;
	char *p;
	int fd;
    fd = open ("lineitem.tbl", O_RDONLY);
	if (fd == -1) {
		perror ("open");
		return 1;
	}
	if (fstat (fd, &sb) == -1) {
		perror ("fstat");
		return 1;
	}
	if (!S_ISREG (sb.st_mode)) {
		fprintf (stderr, "%s is not a file\n", "lineitem.tbl");
		return 1;
	}
	p = (char*)mmap (0, fileSize, PROT_READ, MAP_SHARED, fd, 0);
	if (p == MAP_FAILED) {
		perror ("mmap");
		return 1;
	}
	if (close (fd) == -1) {
		perror ("close");
		return 1;
	}
	thrust::copy(p, p+fileSize, dev.begin());
#endif

    // count lines in file
    int cnt = thrust::count(dev.begin(), dev.end(), '\n');
    std::cout << "There are " << cnt << " total lines in the file" << "\n";

    // char locations for line breaks in vector 
    thrust::device_vector<int> dev_pos(cnt+1);
    dev_pos[0] = -1;
    thrust::copy_if(thrust::make_counting_iterator((unsigned int)0),        // count from start of file
                    thrust::make_counting_iterator((unsigned int)fileSize), // until end of file
                    dev.begin(),        // stencil pred(*stencil)==true causes *dev_pos[] to get location of line break
                    dev_pos.begin()+1,  // position of line break character
                    is_break());        // predicate

    // 11 columns of 15 characters
    thrust::device_vector<char> dev_res1(cnt*15);
    thrust::fill(dev_res1.begin(), dev_res1.end(), 32);
    thrust::device_vector<char> dev_res2(cnt*15);
    thrust::fill(dev_res2.begin(), dev_res2.end(), 32);
    thrust::device_vector<char> dev_res3(cnt*15);
    thrust::fill(dev_res3.begin(), dev_res3.end(), 32);
    thrust::device_vector<char> dev_res4(cnt*15);
    thrust::fill(dev_res4.begin(), dev_res4.end(), 32);
    thrust::device_vector<char> dev_res5(cnt*15);
    thrust::fill(dev_res5.begin(), dev_res5.end(), 32);
    thrust::device_vector<char> dev_res6(cnt*15);
    thrust::fill(dev_res6.begin(), dev_res6.end(), 32);
    thrust::device_vector<char> dev_res7(cnt*15);
    thrust::fill(dev_res7.begin(), dev_res7.end(), 32);
    thrust::device_vector<char> dev_res8(cnt*15);
    thrust::fill(dev_res8.begin(), dev_res8.end(), 32);
    thrust::device_vector<char> dev_res9(cnt);
    thrust::fill(dev_res9.begin(), dev_res9.end(), 32);
    thrust::device_vector<char> dev_res10(cnt);
    thrust::fill(dev_res10.begin(), dev_res10.end(), 32);
    thrust::device_vector<char> dev_res11(cnt*10);
    thrust::fill(dev_res11.begin(), dev_res11.end(), 32);

    // 11 pointers to column data
    thrust::device_vector<char*> dest(11);
    dest[0] = thrust::raw_pointer_cast(dev_res1.data());
    dest[1] = thrust::raw_pointer_cast(dev_res2.data());
    dest[2] = thrust::raw_pointer_cast(dev_res3.data());
    dest[3] = thrust::raw_pointer_cast(dev_res4.data());
    dest[4] = thrust::raw_pointer_cast(dev_res5.data());
    dest[5] = thrust::raw_pointer_cast(dev_res6.data());
    dest[6] = thrust::raw_pointer_cast(dev_res7.data());
    dest[7] = thrust::raw_pointer_cast(dev_res8.data());
    dest[8] = thrust::raw_pointer_cast(dev_res9.data());
    dest[9] = thrust::raw_pointer_cast(dev_res10.data());
    dest[10] = thrust::raw_pointer_cast(dev_res11.data());

    // which field to select / parse
    thrust::device_vector<unsigned int> ind(11); //fields positions
    ind[0] = 0;
    ind[1] = 1;
    ind[2] = 2;
    ind[3] = 3;
    ind[4] = 4;
    ind[5] = 5;
    ind[6] = 6;
    ind[7] = 7;
    ind[8] = 8;
    ind[9] = 9;
    ind[10] = 10;

    // field max length
    thrust::device_vector<unsigned int> dest_len(11); //fields max lengths
    dest_len[0] = 15;
    dest_len[1] = 15;
    dest_len[2] = 15;
    dest_len[3] = 15;
    dest_len[4] = 15;
    dest_len[5] = 15;
    dest_len[6] = 15;
    dest_len[7] = 15;
    dest_len[8] = 1;
    dest_len[9] = 1;
    dest_len[10] = 10;

    // count of fields to parse from each line
    thrust::device_vector<unsigned int> ind_cnt(1); //fields count
    ind_cnt[0] = 10;

    // field separator across line
    thrust::device_vector<char> sep(1);
    sep[0] = '|';

    // split file by line breaks and field separators
    thrust::counting_iterator<unsigned int> begin(0);
    parse_functor ff((const char*)thrust::raw_pointer_cast(dev.data()), // raw file characters
                     (char**)thrust::raw_pointer_cast(dest.data()),     // array of pointers to dest (column) buffers
                     thrust::raw_pointer_cast(ind.data()),              // mapping
                     thrust::raw_pointer_cast(ind_cnt.data()),          // count of columns to parse
                     thrust::raw_pointer_cast(sep.data()),              // separator character
                     thrust::raw_pointer_cast(dev_pos.data()),          // 
                     thrust::raw_pointer_cast(dest_len.data()));
    thrust::for_each(begin, begin + cnt, ff); // now dev_pos vector contains the indexes of new line characters

	std::cout<< "Split out text fields in " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << '\n';

    // parse binary integer results from columns 1-5
    thrust::device_vector<long long int> d_int1(cnt);
    thrust::device_vector<long long int> d_int2(cnt);
    thrust::device_vector<long long int> d_int3(cnt);
    thrust::device_vector<long long int> d_int4(cnt);
    thrust::device_vector<long long int> d_int5(cnt);
    ind_cnt[0] = 15;
    gpu_atoll atoll_ff1( (const char*)thrust::raw_pointer_cast(dev_res1.data()),
                         (long long int*)thrust::raw_pointer_cast(d_int1.data()),
                         thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atoll_ff1);
    gpu_atoll atoll_ff2( (const char*)thrust::raw_pointer_cast(dev_res2.data()),
                         (long long int*)thrust::raw_pointer_cast(d_int2.data()),
                         thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atoll_ff2);
    gpu_atoll atoll_ff3( (const char*)thrust::raw_pointer_cast(dev_res3.data()),
                         (long long int*)thrust::raw_pointer_cast(d_int3.data()),
                         thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atoll_ff3);
    gpu_atoll atoll_ff4( (const char*)thrust::raw_pointer_cast(dev_res4.data()),
                         (long long int*)thrust::raw_pointer_cast(d_int4.data()),
                         thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atoll_ff4);
    gpu_atoll atoll_ff5( (const char*)thrust::raw_pointer_cast(dev_res5.data()),
                         (long long int*)thrust::raw_pointer_cast(d_int5.data()),
                         thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atoll_ff5);
    for(int i = cnt-2; i < cnt; i++)
        std::cout << "Line: " << i << " " << d_int1[i] << " " << d_int2[i] << " " << 
                         d_int3[i] << " " << d_int4[i] << " " << d_int5[i] << "\n";
    std::cout <<  "\n";

    // parse binary double results for columns 6 7 8
    thrust::device_vector<double> d_double6(cnt);
    thrust::device_vector<double> d_double7(cnt);
    thrust::device_vector<double> d_double8(cnt);
    gpu_atof atof_ff6((const char*)thrust::raw_pointer_cast(dev_res6.data()),
                      (double*)thrust::raw_pointer_cast(d_double6.data()),
                      thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atof_ff6);
    gpu_atof atof_ff7((const char*)thrust::raw_pointer_cast(dev_res7.data()),
                      (double*)thrust::raw_pointer_cast(d_double7.data()),
                      thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atof_ff7);
    gpu_atof atof_ff8((const char*)thrust::raw_pointer_cast(dev_res8.data()),
                      (double*)thrust::raw_pointer_cast(d_double8.data()),
                      thrust::raw_pointer_cast(ind_cnt.data()));
    thrust::for_each(begin, begin + cnt, atof_ff8);
    std::cout.precision(10);
    for(int i = cnt-2; i < cnt; i++)
        std::cout << "Line: " << i << " " << d_double6[i] << " " << d_double7[i] << " " << d_double8[i] << "\n";
    std::cout.flush();

    // timing status
	std::cout<< "And parsed 5 x int and 3 x double columns in " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << '\n';

    // dump text results for a few lines - remember that we are dealing with rows rows of 15 characters
    // hence the i*15 + c to calculate the character to (slowly) retreive below.
    for(int i = 0; i < 15; i++) {
        std::cout << "Line " << i << ":";
        for(int c=0; c<15;c++)
            std::cout << dev_res1[i*15+c];
        std::cout << "|";
        for(int c=0; c<15;c++)
            std::cout << dev_res2[i*15+c];
        std::cout << "|";
        for(int c=0; c<15;c++)
            std::cout << dev_res3[i*15+c];
        std::cout << "|";
        for(int c=0; c<15;c++)
            std::cout << dev_res4[i*15+c];
        std::cout << "|";
        for(int c=0; c<15;c++)
            std::cout << dev_res5[i*15+c];
        std::cout << "|";
        for(int c=0; c<15;c++)
            std::cout << dev_res6[i*15+c];
        std::cout << "\n";
    }

    return 0;
}